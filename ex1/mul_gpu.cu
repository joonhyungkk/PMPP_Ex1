#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

// NOTE: if you include stdio.h, you can use printf inside your kernel

#include "common.h"
#include "matrix.h"
#include "mul_gpu.h"
#include "stdio.h"

// TODO (Task 4): Implement matrix multiplication CUDA kernel

__global__ void MatrixMulKernel(float* m, float* n, float* p, int width){
	int row = threadIdx.y * blockDim.y + threadIdx.y;
	int col = threadIdx.x * blockDim.x + threadIdx.x;


	float value = 0.0;
	if( row < width && col<width){
	for(int k = 0; k < width; k++){
		value += m[row * width +k] * n[k * width + col];
		}
	}
	p[row * width + col] = value;

}

void matrix_mul_gpu(const GPUMatrix &m, const GPUMatrix &n, GPUMatrix &p)
{
	// TODO (Task 4): Determine execution configuration and call CUDA kernel
	dim3 dimBlock(p.width, p.height);
	dim3 dimGrid(1,1);
	MatrixMulKernel<<<dimGrid,dimBlock>>>(m.elements,n.elements,p.elements,p.width);
	CUDA_CHECK_ERROR;


}

