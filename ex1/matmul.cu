#include "hip/hip_runtime.h"
#include <iostream>

//#ifdef __HIPCC__
#include <hip/hip_runtime.h>
//#endif

#include "matmul.h"
#include "test.h"
#include "common.h"
#include "mul_cpu.h"
#include "mul_gpu.h"
#include "timer.h"

void print_cuda_devices()
{
	// TODO: Task 2
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);

	for(int i =0; i<count;i++){
		hipGetDeviceProperties(&prop, i);
		std::cout<< "Information for device: " << i <<std::endl;
		std::cout<< "Compute Capability: " << prop.major << "." << prop.minor <<std::endl;
		std::cout<< "Multiprocessor count: " << prop.multiProcessorCount <<std::endl;
		std::cout<< "GPU clock rate: " << prop.clockRate << " GHz" <<std::endl;
		std::cout<< "Total global memory: " << prop.totalGlobalMem << " MiB" <<std::endl;
		std::cout<< "L2 Cache size: " << prop.l2CacheSize << " KiB" <<std::endl;
		std::cout<< "----------------------------------" <<std::endl;

	}

}

void matmul()
{
	// === Task 3 ===
	// TODO: Allocate CPU matrices (see matrix.cc)
	//       Matrix sizes:
	//       Input matrices:
	//       Matrix M: pmpp::M_WIDTH, pmpp::M_HEIGHT
	//       Matrix N: pmpp::N_WIDTH, pmpp::N_HEIGHT
	//       Output matrices:
	//       Matrix P: pmpp::P_WIDTH, pmpp::P_HEIGHT
	CPUMatrix cm = matrix_alloc_cpu(pmpp::M_WIDTH, pmpp::M_HEIGHT);
	CPUMatrix cn = matrix_alloc_cpu(pmpp::N_WIDTH, pmpp::N_HEIGHT);
	CPUMatrix cp = matrix_alloc_cpu(pmpp::P_WIDTH, pmpp::P_HEIGHT);
	
	// TODO: Fill the CPU input matrices with the provided test values (pmpp::fill(CPUMatrix &m, CPUMatrix &n))
	pmpp::fill(cm,cn);

	// TODO (Task 5): Start CPU timing here!
	timer_tp start = timer_now();

	// TODO: Run your implementation on the CPU (see mul_cpu.cc)
	matrix_mul_cpu(cm,cn,cp);

	// TODO (Task 5): Stop CPU timing here!
	timer_tp end = timer_now();
	float elapsed_time = timer_elapsed(start, end);
	printf("CPU proceesing took: %f ms\n", elapsed_time);

	// TODO: Check your matrix for correctness (pmpp::test_cpu(const CPUMatrix &p))
	pmpp::test_cpu(cp);

	// === Task 4 ===
	// TODO: Set CUDA device
	hipSetDevice(0);
	CUDA_CHECK_ERROR;
	std::cout<<"CUDA device setting successful"<<std::endl;
	
	// TODO: Allocate GPU matrices (see matrix.cc)

	CPUMatrix cp2 = matrix_alloc_cpu(pmpp::P_WIDTH, pmpp::P_HEIGHT);
	GPUMatrix gm = matrix_alloc_gpu(pmpp::M_WIDTH, pmpp::M_HEIGHT);
	GPUMatrix gn = matrix_alloc_gpu(pmpp::N_WIDTH, pmpp::N_HEIGHT);
	GPUMatrix gp = matrix_alloc_gpu(pmpp::P_WIDTH, pmpp::P_HEIGHT);

	// TODO: Upload the CPU input matrices to the GPU (see matrix.cc)
	matrix_upload(cm,gm);
	matrix_upload(cn,gn);

	// TODO (Task 5): Start GPU timing here!
	hipEvent_t evStart, evStop;
	hipEventCreate(&evStart);
	hipEventCreate(&evStop);
	hipEventRecord(evStart,0);
	CUDA_CHECK_ERROR;

	// TODO: Run your implementation on the GPU (see mul_gpu.cu)
	matrix_mul_gpu(gm,gn,gp);


	// TODO (Task 5): Stop GPU timing here!
	hipEventRecord(evStop, 0);
	hipEventSynchronize(evStop);
	CUDA_CHECK_ERROR;

	float elapsedTime_ms;
	hipEventElapsedTime(&elapsedTime_ms, evStart, evStop);
	CUDA_CHECK_ERROR;

	printf("CUDA proceesing took: %f ms\n", elapsedTime_ms);
	hipEventDestroy(evStart);
	hipEventDestroy(evStop);
	CUDA_CHECK_ERROR;

	
	// TODO: Download the GPU output matrix to the CPU (see matrix.cc)
	matrix_download(gp,cp2);

	// TODO: Check your downloaded matrix for correctness (pmpp::test_gpu(const CPUMatrix &p))
	pmpp::test_gpu(cp2);

	// TODO: Compare CPU result with GPU result (see matrix.cc)
	matrix_compare_cpu(cp2,cp);

	// TODO (Task3/4/5): Cleanup ALL matrices and and events
	matrix_free_cpu(cm);
	matrix_free_cpu(cn);
	matrix_free_cpu(cp);
	matrix_free_cpu(cp2);
	matrix_free_gpu(gm);
	matrix_free_gpu(gn);
	matrix_free_gpu(gp);
}


/************************************************************
 * 
 * TODO: Write your text answers here!
 * 
 * (Task 4) 6. Where do the differences come from?
 * 
 * Answer: TODO
 * 
 * 
 ************************************************************/
