#include <cstddef>
#include <iostream>
#include <iomanip>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "common.h"
#include "matrix.h"


CPUMatrix matrix_alloc_cpu(int width, int height)
{
	CPUMatrix m;
	m.width = width;
	m.height = height;
	m.elements = new float[m.width * m.height];
	return m;
}
void matrix_free_cpu(CPUMatrix &m)
{
	delete[] m.elements;
}

GPUMatrix matrix_alloc_gpu(int width, int height)
{
	// TODO (Task 4): Allocate memory at the GPU
	GPUMatrix gm;
	gm.width = width;
	gm.height = height;
	gm.elements = (float *)malloc(width*height*sizeof(float)) ;
	hipMallocPitch(&gm.elements, &gm.pitch, width, height);

	CUDA_CHECK_ERROR;
	return gm;

}
void matrix_free_gpu(GPUMatrix &m)
{
	// TODO (Task 4): Free the memory
	hipFree(m.elements);
	CUDA_CHECK_ERROR;

}

void matrix_upload(const CPUMatrix &src, GPUMatrix &dst)
{
	// TODO (Task 4): Upload CPU matrix to the GPU
	int size = src.width*src.height*sizeof(float);
	hipMemcpy2D(dst.elements, dst.pitch, src.elements, size, src.width, src.height, hipMemcpyHostToDevice);
	CUDA_CHECK_ERROR;


}
void matrix_download(const GPUMatrix &src, CPUMatrix &dst)
{
	// TODO (Task 4): Download matrix from the GPU
	int size = dst.width*dst.height*sizeof(float);
	hipMemcpy2D(dst.elements, size, src.elements, src.pitch, dst.width, dst.height, hipMemcpyDeviceToHost);
	CUDA_CHECK_ERROR;

}

void matrix_compare_cpu(const CPUMatrix &a, const CPUMatrix &b)
{
	// TODO (Task 4): compare both matrices a and b and print differences to the console
	for(int i = 0; i <a.width*a.height ;i++){
		if(a.elements[i]!= b.elements[i])
			std::cout<< "Difference detected. "<<i<< " is different. " << a.elements[i] << "and " << b.elements[i] << " are different" <<std::endl;
		else
			continue;
	}
}
		
